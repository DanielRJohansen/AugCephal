#include "hip/hip_runtime.h"
#include "Preprocessing.cuh"

/*void checkCudaError() {
    hipError_t err = hipGetLastError();        // Get error code
    if (err != hipSuccess)
    {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        exit(-1);
    }
}*/








void Preprocessor::insertImInVolume(cv::Mat img, int z) {
    for (int y = 0; y < input_size.y; y++) {
        for (int x = 0; x < input_size.x; x++) {
            float hu = img.at<uint16_t>(y, x) - 32768.; // Load as float here, as we do all further calcs on floats in GPU
            raw_scan[xyzToIndex(Int3(x,y,z), input_size)] = hu;
        }
    }
}

void Preprocessor::loadScans(string folder_path) {
    int successes = 0;
    stringvec v;
    printf("Reading directory %s\n", folder_path.c_str());
    read_directory(folder_path, v);
    for (int i = 2; i < input_size.z + 2; i++) {
        string im_path = folder_path;
        im_path.append(v[i]);
        printf("Loading slice: %s               \r", im_path.c_str());

        cv::Mat img = imread(im_path, cv::IMREAD_UNCHANGED);
        int z = input_size.z - 1 - i + 2;
        if (img.empty()) {
            cout << "\n        Failed!\n" << endl;
            return;
        }
        else successes++;
        insertImInVolume(img, z);
    }

    printf("\n%d Slices loaded succesfully\n", successes);
}


/*__global__ void conversionKernel(Voxel* voxels, float* hu_vals, Int3 size) {
    int x = blockIdx.x;
    int y = threadIdx.x;
    for (int z = 0; z < size.z; z++) {
        int index = xyzToIndex(Int3(x, y, z), size);
        voxels[index].hu_val = 600;
    }
}*/

Volume* Preprocessor::convertToVolume(float* scan, Int3 size) {
    auto start = chrono::high_resolution_clock::now();
    int len = size.x * size.y * size.z;
    unsigned int bytesize = len * sizeof(Voxel);

    // Initialize voxels
    Voxel* v_host = new Voxel[len];
    for (int i = 0; i < len; i++)
        v_host[i].hu_val = scan[i];
    
    // Move voxels to GPU
    Voxel* v_device;
    hipMallocManaged(&v_device, bytesize);
    hipMemcpy(v_device, v_host, bytesize, hipMemcpyHostToDevice);
    printf("%d MB of VRAM allocated to Voxels\n", bytesize / 1000000);

    // C
    Volume* volume = new Volume(v_device, size);

    delete(scan, v_host);

    printf("CPU side conversion in %d ms.\n", chrono::duration_cast<chrono::milliseconds>(chrono::high_resolution_clock::now() - start));
    return volume;
}



__global__ void setIgnoreBelowKernel(Voxel* voxels, float below, Int3 size) {
    int x = blockIdx.x;
    int y = threadIdx.x;
    for (int z = 0; z < size.z; z++) {
        int index = xyzToIndex(Int3(x, y, z), size);
        if (voxels[index].hu_val < below)
            voxels[index].ignore = true;
    }
}
void Preprocessor::setIgnoreBelow(Volume* volume, float below) {
    Int3 size = volume->size;
    setIgnoreBelowKernel << < size.y, size.x >> > (volume->voxels, below, size);
    hipDeviceSynchronize();
}

float* makeNormvalCopy(Volume* vol) {
    Voxel* hostvoxels = new Voxel[vol->len];
    hipMemcpy(hostvoxels, vol->voxels, vol->len * sizeof(Voxel), hipMemcpyDeviceToHost);
    float* copy_host = new float[vol->len];
    for (int i = 0; i < vol->len; i++)
        copy_host[i] = hostvoxels[i].norm_val;

    float* copynorms;
    hipMallocManaged(&copynorms, vol->len * sizeof(float));
    hipMemcpy(copynorms, copy_host, vol->len * sizeof(float), hipMemcpyHostToDevice);
    return copynorms;
}

__global__ void setColumnIgnoresKernel(Voxel* voxels, bool* xyColumnIgnores, Int3 size) {
    int x = blockIdx.x;
    int y = threadIdx.x; 
    int ignore_index = y * size.x + x;
    xyColumnIgnores[ignore_index] = 0;
    int counts = 0;
    for (int z = 0; z < size.z; z++) {
        int index = xyzToIndex(Int3(x, y, z), size);
        if (!voxels[index].ignore) {
            return;
        }            
    }
    xyColumnIgnores[ignore_index] = 1;
}

void Preprocessor::setColumnIgnores(Volume* volume) {
    Int3 size = volume->size;
    int column_len = size.x * size.y;
    int boolbytesize = column_len * sizeof(bool);



    
    hipMallocManaged(&volume->xyColumnIgnores, boolbytesize);

    printf("Allocating xy ignore table of size: %d Kb\n", boolbytesize / 1000);

    setColumnIgnoresKernel << < size.y, size.x >> > (volume->voxels, volume->xyColumnIgnores, size);
    hipDeviceSynchronize();



    volume->CB = new CompactBool(volume->xyColumnIgnores, column_len);

}




__global__ void colorFromNormvalKernel(Voxel* voxels, Int3 size) {
    int x = blockIdx.x;
    int y = threadIdx.x;
    for (int z = 0; z < size.z; z++) {
        int index = xyzToIndex(Int3(x, y, z), size);
        voxels[index].color = CudaColor(voxels[index].norm_val);
    }
}

void Preprocessor::colorFromNormval(Volume* volume) {
    Int3 size = volume->size;
    colorFromNormvalKernel << < size.y, size.x >> > (volume->voxels, size);
    hipDeviceSynchronize();
}














__global__ void windowKernel(Voxel* voxels, float min, float max, Int3 size) {
    int x = blockIdx.x;
    int y = threadIdx.x;
    for (int z = 0; z < size.z; z++) {
        int index = xyzToIndex(Int3(x, y, z), size);
        voxels[index].norm(min, max);
    }
}

void Preprocessor::windowVolume(Volume* volume, float min, float max) {
    auto start = chrono::high_resolution_clock::now();
    Int3 size = volume->size;
    windowKernel << < size.y, size.x >> > (volume->voxels, min, max, size);

    hipError_t err = hipGetLastError();        // Get error code
    if (err != hipSuccess)
    {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        exit(-1);
    }

    hipDeviceSynchronize();
    printf("Windowing executed in %d ms.\n", chrono::duration_cast<chrono::milliseconds>(chrono::high_resolution_clock::now() - start));
}






















__host__ __device__ void makeMasks(CudaMask* masks) {
    int i = 0;
    for (int zs = 0; zs < 3; zs++) {
        for (int ys = 0; ys < 3; ys++) {
            for (int xs = 0; xs < 3; xs++) {
                masks[i] = CudaMask(xs, ys, zs);
                i++;
            }
        }
    }
    
    for (int i = 0; i < 3; i++) {
        masks[i] = CudaMask();
    }
    i = 0;
    for (int z = 0; z < 5; z++) {
        for (int y = 0; y < 5; y++) {
            for (int x = 0; x < 5; x++) {
                //Flats
                masks[27 + 0].mask[i] = (z == 2);
                masks[27 + 1].mask[i] = (y == 2);
                masks[27 + 2].mask[i] = (x == 2);

                // Pillars
                masks[27 + 3].mask[i] = (z == 2 && y == 2);
                masks[27 + 4].mask[i] = (y == 2 && x == 2);
                masks[27 + 5].mask[i] = (x == 2 && z == 2);

                // Semi crooked pillars
                masks[27 + 6].mask[i] = (z == 2 && x == y);
                masks[27 + 7].mask[i] = (z == 2 && x == -y);
                masks[27 + 8].mask[i] = (y == 2 && z == x);
                masks[27 + 9].mask[i] = (y == 2 && z == -x);
                masks[27 + 10].mask[i] = (x == 2 && z == y);
                masks[27 + 11].mask[i] = (x == 2 && z == -y);

                // Full crooked pillars
                masks[27 + 12].mask[i] = (x == y && y == z);
                masks[27 + 13].mask[i] = (x == y && y == -z);
                masks[27 + 14].mask[i] = (-x == y && y == z);
                masks[27 + 15].mask[i] = (-x == y && y == -z);
                /*masks[27 + 3].mask[i] = (z == 0 && abs(x) - 1 <= 0 && abs(y) - 1 <= 0);
                masks[27 + 4].mask[i] = (y == 0 && abs(x) - 1 <= 0 && abs(z) - 1 <= 0);
                masks[27 + 5].mask[i] = (x == 0 && abs(y) - 1 <= 0 && abs(z) - 1 <= 0);*/

                i++;
            }
        }
    }
}

__device__ void fetchWindow(Voxel* voxelcopy, float* kernel, Int3 pos, Int3 size) {
    int i = 0;
    for (int z_ = pos.z - 2; z_ <= pos.z + 2; z_++) {
        for (int y_ = pos.y - 2; y_ <= pos.y + 2; y_++) {
            for (int x_ = pos.x - 2; x_ <= pos.x + 2; x_++) {
                Int3 pos_ = Int3(x_, y_, z_);
                if (pos.z > 0 && z_ < pos.z + 2)
                    kernel[i] = kernel[i + 25];
                else if (!isInVolume(pos_, size))
                    kernel[i] = OUTSIDEVOL;
                else
                {
                    if (voxelcopy[xyzToIndex(pos_, size)].ignore)
                        kernel[i] = ISIGNORE;
                    else
                        kernel[i] = voxelcopy[xyzToIndex(pos_, size)].norm_val;
                }
                i++;
            }
        }
    }
}

__global__ void rotatingMaskFilterKernel(Voxel* voxels, Voxel* voxelcopy, Int3 size, unsigned* ignores, CudaMask* globalmasks) {
    int y = blockIdx.x;  
    int x = threadIdx.x;
    
    CompactBool CB;
    if (CB.getBit(ignores, y * size.x + x)) {
        voxels[xyzToIndex(Int3(x, y, 0), size)].norm_val = 1;
        return;
    }

    // Initialize masks
    CudaMask masks_init[43];
    CudaMask* masks = masks_init;
    for (int i = 0; i < 43; i++)
        masks[i] = globalmasks[i];

    float kernel_[5 * 5 * 5];
    float* kernel = kernel_;
    for (int z = 0; z < size.z; z++) {
        Int3 coord = Int3(x, y, z);
        
        fetchWindow(voxelcopy, kernel, coord, size);        // ALSO DO; EVEN IF WE IGNORE CURRENT VOXEL

        if (voxels[xyzToIndex(coord, size)].ignore)
            continue;

        float best_mean = -1;// voxels[xyzToIndex(coord, size)].norm_val;
        float lowest_var = 999;
        for (int i = 30; i < 43; i++) {
            float var = masks[i].applyMask(kernel);
            if (var < lowest_var) {
                lowest_var = var;
                best_mean = masks[i].mean;
            }
        }
        if (best_mean != -1)
            voxels[xyzToIndex(coord, size)].norm_val = best_mean;  
    }
}

void Preprocessor::rmf(Volume* vol) {
    auto start = chrono::high_resolution_clock::now();

    Voxel* voxelcopy;
    hipMallocManaged(&voxelcopy, vol->len * sizeof(Voxel));
    hipMemcpy(voxelcopy, vol->voxels, vol->len * sizeof(Voxel), hipMemcpyDeviceToDevice);
    //printf("Copy made in %d ms.\n", chrono::duration_cast<chrono::milliseconds>(chrono::high_resolution_clock::now() - start));


    CudaMask* masks = new CudaMask[43];
    CudaMask* gpu_masks;
    makeMasks(masks);
    hipMallocManaged(&gpu_masks, 43 * sizeof(CudaMask));
    hipMemcpy(gpu_masks, masks, 43 * sizeof(CudaMask), hipMemcpyHostToDevice);

    rotatingMaskFilterKernel << <vol->size.y, vol->size.x >> > (vol->voxels, voxelcopy, vol->size, vol->CB->compact_gpu, gpu_masks);
    hipDeviceSynchronize();
    
    hipFree(voxelcopy);
    hipFree(gpu_masks);
    delete(masks);
    printf("RMF applied in %d ms.\n", chrono::duration_cast<chrono::milliseconds>(chrono::high_resolution_clock::now() - start));
}




















//----------------------------------------------------------------------Clustering----------------------------------------------------------------------------------------------------\\







const int x_off[6] = { 0, 0, 0, 0, -1, 1 };
const int y_off[6] = { 0, 0, -1, 1, 0, 0 };
const int z_off[6] = { -1, 1, 0, 0, 0, 0 };

void propagateCluster(Volume* vol, TissueCluster3D* cluster, Int3 pos, int depth) {
    //if (!(depth%1000)) printf("depth: %d\r", depth);
    
    int index = xyzToIndex(pos, vol->size);
    vol->voxels[index].cluster_id = cluster->id;
    cluster->addMember(index);
    // load val into cluster accumulation here to save time

    for (int i = 0; i < 6; i++) {
        int x_ = pos.x + x_off[i];
        int y_ = pos.y + y_off[i];
        int z_ = pos.z + z_off[i];
        Int3 pos_(x_, y_, z_);

        if (isInVolume(pos_, vol->size)) {
            int index_ = xyzToIndex(pos_, vol->size);
            Voxel voxel = vol->voxels[index_];
            if (!voxel.ignore) {
                if (voxel.cluster_id == -1 && voxel.kcluster == cluster->target_kcluster)
                    propagateCluster(vol, cluster, pos_, depth + 1);
            }
        }
    }
}


void clusterInitializationTask(TissueCluster3D* cluster, Volume* vol) {
    cluster->initialize(vol);

}

void clusterInitScheduler(vector<TissueCluster3D*> clusters, Volume* vol) {
    vector<thread> workers;
    unsigned int edge_voxels = 0;
    for (int i = 0; i < clusters.size(); i++) {
        char index = 0;
        if (clusters[i]->member_indexes.size() < 500000) {
            clusters[i]->initialize(vol);
        }
        else {
            thread worker(clusterInitializationTask, clusters[i], vol);
            workers.push_back(move(worker));
        }
    }
    printf("\n");
    for (int i = 0; i < workers.size(); i++) {
        printf("\rWaiting to join threads (%02d/%02d)  ", i+1, workers.size());
        workers[i].join();
    }
    printf("\n");
}

void testTask(bool* available_threads, char thread_index) {
    int a = 0;
    available_threads[thread_index] = 1;

}


vector<TissueCluster3D*> Preprocessor::clusterSync(Volume* vol) {
    printf("Clustering initiated...");
    auto start = chrono::high_resolution_clock::now();

    Int3 size = vol->size;
    int id = 0;
    CudaColor color = CudaColor().getRandColor();
    vector<TissueCluster3D*> clusters;
    

    for (int z = 0; z < size.z; z++) {
        for (int y = 0; y < size.y; y++) {
            for (int x = 0; x < size.x; x++) {
                Int3 pos(x, y, z);
                int index = xyzToIndex(pos, size);
                Voxel voxel = vol->voxels[index];
                if (voxel.cluster_id == -1 && !voxel.ignore) {
                    TissueCluster3D* cluster = new TissueCluster3D(id, voxel.kcluster);
                    propagateCluster(vol, cluster, Int3(x, y, z), 0);

                    clusters.push_back(cluster);
                    id++;
                }
            }
        }
    }



    auto t1 = chrono::high_resolution_clock::now();
    printf("  %d clusters found in %d ms \n", id, chrono::duration_cast<chrono::milliseconds>(t1 - start));


    clusterInitScheduler(clusters, vol);
    printf("Clusters initialized in %d ms!\n\n", chrono::duration_cast<chrono::milliseconds>(chrono::high_resolution_clock::now() - t1));
    return clusters;
}

int* bucketSort(int* sizes, int num) {	// Fucks up on size 0 or less!
    int* ordered_indexes = new int[num];
    int head = num - 1;

    int bucket_start = 0;
    int bucket_end = 4;
    while (true) {
        for (int i = 0; i < num; i++) {
            int sizei = sizes[i];
            if (sizei >= bucket_start && sizei < bucket_end) {
                ordered_indexes[head] = i;
                head--;
                if (head == -1)
                    return ordered_indexes;
            }
        }
        bucket_start = bucket_end;
        bucket_end *= 2;
        if (bucket_end == 0) {
            printf("SOMETHING WENT WRONG");
            break;
        }

    }
}
int* orderClustersBySize(vector<TissueCluster3D*> clusters) {
    int* sizes = new int[clusters.size()];
    for (int i = 0; i < clusters.size(); i++) {
        sizes[i] = clusters[i]->getSize();
    }
    int* ordered_indexes = bucketSort(sizes, clusters.size());
    delete(sizes);
    return ordered_indexes;
}


int Preprocessor::mergeClusters(Volume* vol, vector<TissueCluster3D*> clusters, int remaining_clusters) {
    auto start = chrono::high_resolution_clock::now();


    for (int i = 0; i < clusters.size(); i++) {
        clusters[i]->viable_neighbor_ids.copy(clusters[i]->neighbor_ids);
    }


    int* ordered_index = orderClustersBySize(clusters);
    for (int i = 0; i < clusters.size(); i++) {
        clusters[ordered_index[i]]->mergeClusters(&clusters);
    }
    delete(ordered_index);

    printf("\nMerging completed in %d ms!\n", chrono::duration_cast<chrono::milliseconds>(chrono::high_resolution_clock::now() - start));
    return countAliveClusters(clusters, remaining_clusters);
}

int Preprocessor::eliminateVesicles(Volume* vol, vector<TissueCluster3D*> clusters, int threshold_size, int remaining_clusters) {
    auto start = chrono::high_resolution_clock::now();
    printf("Removing vesicles...");

    for (int i = 0; i < clusters.size(); i++) {
        clusters[i]->eliminateVesicle(vol, &clusters, threshold_size);
    }

    printf("    completed in %d ms!\n", chrono::duration_cast<chrono::milliseconds>(chrono::high_resolution_clock::now() - start));
    return countAliveClusters(clusters, remaining_clusters);
}


void Preprocessor::finalizeClusters(Volume* vol, vector<TissueCluster3D*> clusters) {
    auto start = chrono::high_resolution_clock::now();
    ColorMaker CM;
    for (int i = 0; i < clusters.size(); i++) {
        clusters[i]->finalize(vol, &CM);
    }

    printf("Clusters finalized in %d ms!\n", chrono::duration_cast<chrono::milliseconds>(chrono::high_resolution_clock::now() - start));
}

int Preprocessor::countAliveClusters(vector<TissueCluster3D*> clusters, int from) {
    int num_dead = 0;
    for (int i = 0; i < clusters.size(); i++) {
        if (clusters[i]->dead)
            num_dead++;
    }
    int reduced = from - (clusters.size()-num_dead);
    printf("Num clusters reduced by %d          %d->%d\n", reduced, from, clusters.size() - num_dead);
    return from - reduced;
}










TissueCluster3D* Preprocessor::removeExcessClusters(vector<TissueCluster3D*> clusters, int remaining_clusters) {
    TissueCluster3D* compressed_clusters = new TissueCluster3D[remaining_clusters];
    int compressed_index = 0;
    for (int i = 0; i < clusters.size(); i++) {
        if (!clusters[i]->dead) {
            compressed_clusters[compressed_index].copyMinInfo(clusters[i]);
            compressed_clusters[compressed_index].id = compressed_index;
            compressed_index++;
        }
        clusters[i]->empty();
    }
    clusters.clear();
    printf("Clustervector compressed to %d clusters\n", compressed_index);
    return compressed_clusters;
}


int* generateClusterIDMap(vector<TissueCluster3D*> clusters, int remaining_clusters) {
    int* map = new int[clusters.size()];
    int compact_index = 0;
    for (int i = 0; i < clusters.size(); i++) {
        if (!clusters[i]->dead) {
            map[i] = compact_index;
            compact_index++;
        }
        else
            map[i] = -1;
    }
    assert(compact_index == remaining_clusters);
    return map;
}

RenderVoxel* Preprocessor::compressVoxels(Volume* vol, vector<TissueCluster3D*> clusters, int remaining_clusters) {
    
    RenderVoxel* rvoxels = new RenderVoxel[vol->len];
    int* clustermap = generateClusterIDMap(clusters, remaining_clusters);
    
    for (int i = 0; i < vol->len; i++) {
        //if (!vol->voxels[i].ignore)
        if (vol->voxels[i].isEdge && !vol->voxels[i].ignore)
            rvoxels[i].cluster_id = clustermap[vol->voxels[i].cluster_id];
        else
            rvoxels[i].cluster_id = -1;
    }


    RenderVoxel* rvoxels_dev;
    int bytesize = vol->len * sizeof(RenderVoxel);
    printf("Allocating rendervoxels on gpu: %d MB", bytesize / 1000000);
    hipMallocManaged(&rvoxels_dev, bytesize);
    hipMemcpy(rvoxels_dev, rvoxels, bytesize, hipMemcpyHostToDevice);

    hipFree(vol->voxels);
    delete(rvoxels);
    delete(clustermap);
    return rvoxels_dev;
}






