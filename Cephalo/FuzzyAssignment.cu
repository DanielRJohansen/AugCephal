#include "hip/hip_runtime.h"
#include "FuzzyAssignment.cuh"

//------------------------------------------------------------------------------------------------------------------K MEANS --------------------------------------------------------------------------------//

__managed__ float kcluster_total_change = 99;

//--------------------------KERNEL Helper functions----------------------//

__device__ void updateGlobalClustersIntoShared(CudaKCluster* shared_clusters, CudaKCluster* kclusters, int k) {
    int x = threadIdx.x;

    if (x < k) {
        shared_clusters[x] = kclusters[x];
    }
}

__device__ void resetAccumulations(float* thread_accs, short* thread_mems, int k) {    // accs->accumulations, mems->num. members  
    int x = threadIdx.x;
    int offset = x * k;
    for (int i = 0; i < k; i++) {
        thread_accs[offset + i] = 0;
        thread_mems[offset + i] = 0;
    }
}

__device__ void updateSharedMemClusters(CudaKCluster* shared_clusters, float* thread_accs, short* thread_mems, int k, int threads_per_block) {      // Here it is vital we sync, both before and after updating!
    int x = threadIdx.x;
    __syncthreads();
    if (x < k) {
        for (int i = 0; i < threads_per_block; i++) {
            int index = x + i * k;
            shared_clusters[x].assignBatch(thread_accs[index], thread_mems[index]);
        }
    }
    __syncthreads();
}

__device__ void pushSharedMemClusterToGlobalBlockClusters(CudaKCluster* shared_clusters, CudaKCluster* global_clusters, int k) {
    int y = blockIdx.x;
    int x = threadIdx.x;
    if (x < k)
        global_clusters[y * k + x] = shared_clusters[x];
    __syncthreads();                                                    // Only relevant if we do NOT exit kernel afterwards
}
__device__ void resetBelongings(float* belongings, int k) {
    for (int i = 0; i < k; i++) {
        belongings[i] = 0;
    }
}

__device__ int getBestBelongingIndex(float* belongings, int k) {
    int best_index = 0;
    float best_belonging = 0;
    for (int i = 0; i < k; i++) {
        if (belongings[i] > best_belonging) {
            best_belonging = belongings[i];
            best_index = i;
        }
    }
    return best_index;
}


__device__ void fetchWindow3x3(Voxel* voxelcopy, float* kernel, Int3 pos, Int3 size) {
    int i = 0;
    for (int z_ = pos.z - 1; z_ <= pos.z + 1; z_++) {
        for (int y_ = pos.y - 1; y_ <= pos.y + 1; y_++) {
            for (int x_ = pos.x - 1; x_ <= pos.x + 1; x_++) {
                Int3 pos_ = Int3(x_, y_, z_);
                if (pos.z > 0 && z_ < pos.z + 1 && false)
                    kernel[i] = kernel[i + 9];
                else if (!isInVolume(pos_, size))
                    kernel[i] = OUTSIDEVOL;
                else
                {
                    if (voxelcopy[xyzToIndex(pos_, size)].ignore)
                        kernel[i] = ISIGNORE;
                    else
                        kernel[i] = voxelcopy[xyzToIndex(pos_, size)].norm_val;
                }
                i++;
            }
        }
    }
}


//--------------------------Kernels----------------------//
__global__ void kMeansRunKernel(Voxel* voxels, CudaKCluster* kclusters, CudaKCluster* global_clusters, int k, Int3 size, int threads_per_block) {
    int index = blockIdx.x *threads_per_block + threadIdx.x;
    int y = index / size.x;
    int x = index % size.x;
    if (y >= size.y || x >= size.x)     // May happen in the very last block
        return;

    extern __shared__ CudaKCluster block_clusters[];
    CudaKCluster* shared_clusters = (CudaKCluster*)block_clusters;      // k per block
    float* thread_accs = (float*)&shared_clusters[k];                   // k * num_threads per block
    short* thread_mems = (short*)&thread_accs[k * threads_per_block];

    updateGlobalClustersIntoShared(shared_clusters, kclusters, k);  // Load clusters into shared mem  
    resetAccumulations(thread_accs, thread_mems, k);                // Init thread mem
    __syncthreads();

    // Algo
    int thread_offset = threadIdx.x * k;
    for (int z = 0; z < size.z; z++) {
        Voxel voxel = voxels[xyzToIndex(Int3(x, y, z), size)];
        if (!voxel.ignore) {
            float highest_belonging = 0;
            int best_index = 0;

            for (int i = 0; i < k; i++) {
                float belonging = shared_clusters[i].belonging(voxel.norm_val);
                if (belonging > highest_belonging) {
                    highest_belonging = belonging;
                    best_index = i;
                }
            }
            int thread_k_index = thread_offset + best_index;
            thread_accs[thread_k_index] += voxel.norm_val;
            thread_mems[thread_k_index] += 1;
        }
    }

    updateSharedMemClusters(shared_clusters, thread_accs, thread_mems, k, threads_per_block);
    pushSharedMemClusterToGlobalBlockClusters(shared_clusters, global_clusters, k);
}
/*
__global__ void kMeansRunKernel2(Voxel* voxels, CudaKCluster* kclusters, CudaKCluster* global_clusters, int k, Int3 size) {
    int y = blockIdx.x;
    int x = threadIdx.x;

    extern __shared__ CudaKCluster block_clusters[];
    CudaKCluster* shared_clusters = (CudaKCluster*)block_clusters;
    float* thread_accs = (float*)&shared_clusters[k];
    short* thread_mems = (short*)&thread_accs[k * size.x];

    updateGlobalClustersIntoShared(shared_clusters, kclusters, k);  // Load clusters into shared mem  
    resetAccumulations(thread_accs, thread_mems, k);                // Init thread mem

    // Algo
    int offset = x * k;
    for (int z = 0; z < size.z; z++) {
        Voxel voxel = voxels[xyzToIndex(Int3(x, y, z), size)];
        if (!voxel.ignore) {
            float highest_belonging = 0;
            int best_index = 0;

            for (int i = 0; i < k; i++) {
                float belonging = shared_clusters[i].belonging(voxel.norm_val);
                if (belonging > highest_belonging) {
                    highest_belonging = belonging;
                    best_index = i;
                }
            }
            int index = offset + best_index;
            thread_accs[index] += voxel.norm_val;
            thread_mems[index] += 1;
        }
    }

    updateSharedMemClusters(shared_clusters, thread_accs, thread_mems, k, size);
    pushSharedMemClusterToGlobalBlockClusters(shared_clusters, global_clusters, k);
}
*/
/*
__global__ void updateGlobalClustersKernel2(CudaKCluster* kclusters, CudaKCluster* block_clusters, int k, Int3 size) { // block_clusters are in global memory
    int x = threadIdx.x;                                        // Which k to handle

    extern __shared__ float change_arr[];
    float* shared_change = (float*)change_arr;

    for (int i = 0; i < size.y; i++) {
        kclusters[x].mergeBatch(block_clusters[i * k + x]);
    }

    shared_change[x] = kclusters[x].calcCentroid();;

    __syncthreads();

    if (x == 0) {
        kcluster_total_change = 0;
        for (int i = 0; i < k; i++)
            kcluster_total_change += shared_change[i];
    }
}
*/

__global__ void updateGlobalClustersKernel(CudaKCluster* kclusters, CudaKCluster* block_clusters, int k, int num_blocks) { // block_clusters are in global memory
    int x = threadIdx.x;

    extern __shared__ float change_arr[];
    float* shared_change = (float*)change_arr;

    for (int i = 0; i < num_blocks; i++) {
        kclusters[x].mergeBatch(block_clusters[i * k + x]);
    }

    shared_change[x] = kclusters[x].calcCentroid();;

    __syncthreads();

    if (x == 0) {
        kcluster_total_change = 0;
        for (int i = 0; i < k; i++)
            kcluster_total_change += shared_change[i];
    }
}



__global__ void fuzzyAssignmentKernel(Voxel* voxels, CudaKCluster* kclusters, float* gauss_kernel, int k, Int3 size) {
    int y = blockIdx.x;
    int x = threadIdx.x;

    int thread_offset = x * k;
    extern __shared__ float block_belongings[];
    float* belongings = (float*)&block_belongings[thread_offset];


    float window[27];

    for (int z = 0; z < size.z; z++) {
        resetBelongings(belongings, k);
        int neighbor_ignores = 0;
        Int3 pos(x, y, z);

        Voxel voxel = voxels[xyzToIndex(pos, size)];

        //fetchWindow3x3(voxels, window, pos, size);  // ALWAYS DO OR THIS SHIT DONT WORK


        if (!voxel.ignore) {
         
            for (int z_ = z - 1; z_ <= z + 1; z_++) {
                for (int y_ = y - 1; y_ <= y + 1; y_++) {
                    for (int x_ = x - 1; x_ <= x + 1; x_++) {
                        if (isInVolume(Int3(x_, y_, z_), size)) {
                            int gauss_kernel_index = (z_ - z) * 9 + (y_ - y) * 3 + (x_ - x);
                            Voxel voxel_ = voxels[xyzToIndex(Int3(x_, y_, z_), size)];
                            if (!voxel_.ignore) {
                                for (int i = 0; i < k; i++) {
                                    belongings[i] += kclusters[i].belonging(voxel_.norm_val) * gauss_kernel[gauss_kernel_index];
                                }
                            }
                            else
                                neighbor_ignores++;
                        }
                    }
                }
            }


            int best_index = getBestBelongingIndex(belongings, k);
            voxel.color = kclusters[best_index].color;
            //voxel.norm_val = kclusters[best_index].centroid;
            voxel.kcluster = best_index;
            if (neighbor_ignores > 20)
                voxel.ignore = true;
            voxels[xyzToIndex(Int3(x, y, z), size)] = voxel;
        }
    }
}








//------------------------------------------------------HOST Helper functions--------------------------------------------------------------------------------------//
void checkCudaError() {
    hipError_t err = hipGetLastError();        // Get error code
    if (err != hipSuccess)
    {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        exit(-1);
    }
}


int inline normvalToHuval(float norm) {return norm * 1500. - 700;}

void printKmeansStuff(CudaKCluster* cluster_dev, int k) {
    CudaKCluster* kc_host = new CudaKCluster[k];
    hipMemcpy(kc_host, cluster_dev, k * sizeof(CudaKCluster), hipMemcpyDeviceToHost);                       // Basically just copies the id
    printf("\n");
    for (int i = 0; i < k; i++) {
        CudaKCluster kc = kc_host[i];
        printf("    K-Cluster %02d	centroid: %05d    members: %d  \n", kc.id, normvalToHuval(kc.centroid), kc.prev_members);
    }
    printf("\n");
}

CudaKCluster* initClusters(int k) {
    CudaKCluster* kclusters_host = new CudaKCluster[k];
    for (int i = 0; i < k; i++)
        kclusters_host[i] = CudaKCluster(i, k);
    CudaKCluster* kclusters_device;
    hipMallocManaged(&kclusters_device, k * sizeof(CudaKCluster));
    hipMemcpy(kclusters_device, kclusters_host, k * sizeof(CudaKCluster), hipMemcpyHostToDevice);
    return kclusters_device;
}

float dist(Int3 o, Int3 p) {
    float x_ = o.x - p.x;
    float y_ = o.y - p.y;
    float z_ = o.z - p.z;
    return sqrt(x_ * x_ + y_ * y_ + z_ * z_);
}

float* makeGaussianKernel3D() {
    float* kernel = new float[3 * 3 * 3];
    Int3 o(0, 0, 0);
    int index = 0;
    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                Int3 p(x, y, z);
                kernel[index++] = 1 / (1 + dist(o, p));
            }
        }
    }
    float* kernel_dev;
    hipMallocManaged(&kernel_dev, 3 * 3 * 3 * sizeof(float));
    hipMemcpy(kernel_dev, kernel, 3 * 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
    delete(kernel);
    return kernel_dev;
}

void checkFuzzyAssignment(Volume* vol, int k) {
    Voxel* vh = new Voxel[vol->len];
    hipMemcpy(vh, vol->voxels, vol->len * sizeof(Voxel), hipMemcpyDeviceToHost);
    int* belongings = new int[k+1]();
    for (int i = 0; i < vol->len; i++) {
        belongings[vh[i].kcluster+1] += 1;
    }
    printf("Fuzzy assignment: \n");
    for (int i = -1; i < k; i++)
        printf("Kluster %d  members: %d\n", i, belongings[i+1]);

}


//---------------------------------------------KERNEL launchers -----------------------------------------------------------------------------------------------------------------//



CudaKCluster* FuzzyAssigner::kMeans(Volume* vol, int k, int max_iterations) {                                    // We must launch separate kernels to update clusters. Only 100% safe way to sync threadblocks!
    auto start = chrono::high_resolution_clock::now();


    int threads_per_block = 128;
    int num_blocks = (vol->size.x * vol->size.y) / threads_per_block;


    CudaKCluster* kclusters_device = initClusters(k);
    CudaKCluster* kclusters_blocks;
    hipMallocManaged(&kclusters_blocks, num_blocks * k * sizeof(CudaKCluster));   // K clusters for each BLOCK

    int shared_mem_size = k * sizeof(CudaKCluster) + k * threads_per_block * sizeof(float) + k * threads_per_block * sizeof(short);
    printf("\n\nExecuting kMeans with %d clusters.\nAllocating %d Kb of memory on %d threadblocks\n", k, shared_mem_size / 1000, num_blocks);




    int iterations = 0;
    while (kcluster_total_change > 0.002 && iterations < max_iterations) {
        
        kMeansRunKernel << <num_blocks, threads_per_block, shared_mem_size >> > (vol->voxels, kclusters_device, kclusters_blocks, k, vol->size, threads_per_block);
        hipDeviceSynchronize();
        checkCudaError();
        
        updateGlobalClustersKernel << <1, k, k * sizeof(float) >> > (kclusters_device, kclusters_blocks, k, num_blocks);
        checkCudaError();
        hipDeviceSynchronize();

        printf("Total change for kclusters: %f    iterations: %02d\r", kcluster_total_change, iterations++);
    }
    printKmeansStuff(kclusters_device, k);


    printf("KCluster found in %d ms.\n", chrono::duration_cast<chrono::milliseconds>(chrono::high_resolution_clock::now() - start));
    return kclusters_device;
}


void FuzzyAssigner::fuzzyClusterAssignment(Volume* vol, CudaKCluster* kclusters_dev, int k) {
    auto start = chrono::high_resolution_clock::now();


    float* gauss_kernel_dev = makeGaussianKernel3D();
    int shared_mem_size = k * vol->size.x * sizeof(float);
    fuzzyAssignmentKernel << <vol->size.y, vol->size.x, shared_mem_size >> > (vol->voxels, kclusters_dev, gauss_kernel_dev, k, vol->size);
    hipDeviceSynchronize();

    //checkFuzzyAssignment(vol, k);

    printf("Fuzzy assignment completed in %d ms.\n\n\n", chrono::duration_cast<chrono::milliseconds>(chrono::high_resolution_clock::now() - start));

}