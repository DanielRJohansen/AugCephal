#include "hip/hip_runtime.h"
#include "CudaOps.cuh"

#define vol_x_range VOL_X
#define vol_y_range VOL_Y
#define vol_z_range VOL_Z
#define RAY_SS 1
//hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);
__global__ void squareKernel(int* a)
{
    int i = blockIdx.x * 512 + threadIdx.x;
    a[i] = i;
}

__global__ void stepKernel(Ray* rayptr, Block *blocks) {
    int index = blockIdx.x * RAYS_PER_DIM + threadIdx.x;  //This fucks shit up if RPD > 1024!!

    //Reset ray
    rayptr[index].acc_color = 0;
    rayptr[index].acc_alpha = 0;
    rayptr[index].full = false;


    float sin_pitch = sin(rayptr[index].cam_pitch);
    float cos_pitch = cos(rayptr[index].cam_pitch);
    float sin_yaw = sin(rayptr[index].cam_yaw);
    float cos_yaw = cos(rayptr[index].cam_yaw);

    float x = rayptr[index].rel_unit_vector.x;
    float y = rayptr[index].rel_unit_vector.y;
    float z = rayptr[index].rel_unit_vector.z;

    float x_y = cos_pitch * x + sin_pitch * z;
    float y_y = y;
    float z_y = -sin_pitch * x + cos_pitch * z;

    // Rotate relative vector about z
    float x_z = cos_yaw * x_y - sin_yaw * y_y;
    float y_z = sin_yaw * x_y + cos_yaw * y_y;
    float z_z = z_y;
    
    float x_ = x_z * RAY_SS;
    float y_ = y_z * RAY_SS;
    float z_ = z_z * RAY_SS;

    for (int step = 20; step < RAY_STEPS; step++) {
        if (rayptr[index].full) {
            break;
        }

        float x = rayptr[index].origin.x + x_ * step;
        float y = rayptr[index].origin.y + y_ * step;
        float z = rayptr[index].origin.z + z_ * step;
        int vol_x = (int)x + vol_x_range / 2;
        int vol_y = (int)y + vol_y_range / 2;
        int vol_z = (int)z + vol_z_range / 2;
                
        if (vol_x >= 0 && vol_y >= 0 && vol_z >= 0 && // Only proceed if coordinate is within volume!
            vol_x < vol_x_range && vol_y < vol_y_range && vol_z < vol_z_range) {
            int volume_index = vol_z * VOL_X * VOL_Y + vol_y * VOL_X + vol_x;
            if (blocks[volume_index].air)
                continue;
            else {
                rayptr[index].acc_color += blocks[volume_index].value;
                rayptr[index].full = true;
            }
            //rayptr[index].acc_alpha += blocks[volume_index].alpha;
     
        }
    }
}


CudaOperator::CudaOperator(){
    hipMallocManaged(&rayptr, NUM_RAYS * sizeof(Ray));
    hipMallocManaged(&blocks, VOL_X*VOL_Y*VOL_Z*sizeof(Block));
    cout << "Cuda initialized" << endl;
    }


void CudaOperator::newVolume(Block* bs) { 
    hipMemcpy(blocks, bs, VOL_X*VOL_Y*VOL_Z * sizeof(Block), hipMemcpyHostToDevice);
    //Volume only needs to go one way as it is not altered.
}


void CudaOperator::rayStep(Ray *rp) {
    hipMemcpy(rayptr, rp, NUM_RAYS * sizeof(Ray), hipMemcpyHostToDevice);
    hipDeviceSynchronize();


    stepKernel << <RAYS_PER_DIM, RAYS_PER_DIM >> > (rayptr, blocks);    // RPD blocks (y), RPD threads(x)
    hipDeviceSynchronize();


    //Finally the CUDA altered rayptr must be copied back to the Raytracer rayptr
    hipMemcpy(rp, rayptr, NUM_RAYS * sizeof(Ray), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
}


