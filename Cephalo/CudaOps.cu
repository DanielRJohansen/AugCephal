#include "hip/hip_runtime.h"
#include "CudaOps.cuh"

#define vol_x_range VOL_X
#define vol_y_range VOL_Y
#define vol_z_range VOL_Z
//hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);
__global__ void squareKernel(int* a)
{
    int i = blockIdx.x * 512 + threadIdx.x;
    a[i] = i;
}

__global__ void stepKernel(Ray* rayptr, Block *blocks, bool *success, float* coor) {
    int index = blockIdx.x * RAYS_PER_DIM + threadIdx.x;  //This fucks shit up if RPD > 1024!!
    

    for (int step = 0; step < 2000; step++) {
        if (true){//!rayptr[index].full) {
            float x_ = rayptr[index].origin.x + rayptr[index].step_vector.x * step;
            float y_ = rayptr[index].origin.y + rayptr[index].step_vector.y * step;
            float z_ = rayptr[index].origin.z + rayptr[index].step_vector.z * step;
            int vol_x = (int)x_ + vol_x_range / 2;
            int vol_y = (int)y_ + vol_y_range / 2;
            int vol_z = (int)z_ + vol_z_range / 2;
            if (index == 0) {
                coor[step] = z_;
            }
                
            if (vol_x >= 0 && vol_y >= 0 && vol_z >= 0 && // Only proceed if coordinate is within volume!
                vol_x < vol_x_range && vol_y < vol_y_range && vol_z < vol_z_range) {
                int volume_index = vol_z * 512 * 512 + vol_y * 512 + vol_x;
                rayptr[index].acc_color += blocks[volume_index].color;
                rayptr[index].acc_alpha += blocks[volume_index].alpha;
                if (rayptr[index].acc_alpha >= 1) 
                    rayptr[index].full = true;
            }
        }
    }
    * success = 1;
}


CudaOperator::CudaOperator(){
    //cout << (NUM_RAYS * sizeof(Ray))/1000000. << " MB" << "  " << sizeof(Block)/1000000. << " MB" <<endl;
    hipMallocManaged(&rayptr, NUM_RAYS * sizeof(Ray));
    hipMallocManaged(&t, 50 * sizeof(testObject));
    hipMallocManaged(&blocks, 512*512*30*sizeof(Block));
    cout << "Cuda initialized" << endl;
    }
void CudaOperator::newVolume(Block* bs) { 
    hipMemcpy(blocks, bs, 512*512*30 * sizeof(Block), hipMemcpyHostToDevice);
    //Volume only needs to go one way as it is not altered.
}


void CudaOperator::rayStep(Ray *rp) {
    hipMemcpy(rayptr, rp, NUM_RAYS * sizeof(Ray), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    bool* success;
    hipMallocManaged(&success, sizeof(bool));
    *success = false;
    float* coor;
    hipMallocManaged(&coor, 2000*sizeof(float));

    stepKernel << <RAYS_PER_DIM, RAYS_PER_DIM >> > (rayptr, blocks, success, coor);    // RPD blocks (y), RPD threads(x)
    hipDeviceSynchronize();
    cout << "Success: " << *success << endl;
    cout << rayptr[0].origin.y << endl;
    for (int i = 0; i < 100; i++) {
        //cout << rayptr[i].acc_color << endl;
        cout << coor[i] << " ";
    }
    cout << endl;
    //cout << rayptr[120].acc_color << "  " << rayptr[140].acc_color;
    cout << "Raystep complete "<< endl;

    //Finally the CUDA altered rayptr must be copied back to the Raytracer rayptr
    hipMemcpy(rp, rayptr, NUM_RAYS * sizeof(Ray), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
}
















void CudaOperator::doStuff() {
    int* a;
    hipMallocManaged(&a, 50*512 * sizeof(int));

    squareKernel << <50, 512 >> > (a);
    hipDeviceSynchronize();
    for (int i = 0; i < 50*512; i++) {
        cout << a[i] << " ";
    }
    hipFree(a);
}

__global__ void testKernel(testObject *t, float* a, bool *finished)
{
    int i = threadIdx.x;
    a[i] = t[i].var*2;
    t[i].var = t[i].var / 2;
    *finished = true;
}
void CudaOperator::objectTesting(testObject* te) {
    //t = te;
    hipMemcpy(t, te, 50 * sizeof(testObject), hipMemcpyHostToDevice);
    cout << t[0].var << endl;
    float* a;
    hipMallocManaged(&a, 50 * sizeof(float));

    bool* finished;
    hipMallocManaged(&finished, sizeof(bool));
    *finished = false;
    testKernel << <1, 50 >> > (t, a, finished);
    hipDeviceSynchronize();
    cout <<"Finished: " << *finished << endl;
    for (int i = 0; i < 50; i++) {
        cout << t[i].var << " ";
    }
    cout << endl;
    hipMemcpy(te, t, 50 * sizeof(testObject), hipMemcpyDeviceToHost);


}