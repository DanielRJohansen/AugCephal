#include "hip/hip_runtime.h"
#include "CudaContainers.cuh"

Int3::Int3(CudaFloat3 s) { x = s.x; y = s.y; z = s.z; }






void TissueCluster3D::mergeClusters(Volume* vol, TissueCluster3D* all_clusters) {	// you know, because it has no parent yet
	return;

	int* ids = neighbor_ids.fetch();
	for (int i = 0; i < neighbor_ids.size(); i++) {

		TissueCluster3D* neighbor = &all_clusters[ids[i]];

		if (isMergeable(neighbor)) {
			if (member_indexes.size() > neighbor->member_indexes.size()) {
				mergeCluster(vol->voxels, neighbor);
			}
			else {
				neighbor->mergeCluster(vol->voxels, &all_clusters[id]);
				neighbor->mergeClusters(vol, all_clusters);		// Continue merging on the new parent, who now owns all of this clusters' neighbors
				delete(ids);
				return;												// Cannot continue as new parent have deleted everything if this cluster.
			}
		}
	}
	delete(ids);
}


void TissueCluster3D::findNeighborsAndMean(Volume* vol) {
	for (int i = 0; i < n_members; i++) {
		int member_index = member_indexes[i];
		Voxel* cur_voxel = &vol->voxels[member_index];	// No ignores are added to a cluster
		mean += (double)cur_voxel->hu_val / (double)n_members;


		Int3 origin = indexToXYZ(member_index, vol->size);

		// search each neighbor-voxel for neighbor cluster
		for (int i = 0; i < 6; i++) {
			Int3 pos = getImmediateNeighbor(origin, i);
			if (isInVolume(pos, vol->size)) {
				Voxel* neighbor = &vol->voxels[xyzToIndex(pos, vol->size)];
				if (neighbor->cluster_id != cur_voxel->cluster_id) {
					neighbor_ids.addVal(neighbor->cluster_id);
				}
			}
		}
	}
}




void TissueCluster3D::mergeCluster(Voxel* voxels, TissueCluster3D* orphan) {	// you know, because it has no parent yet

	mean = (mean * n_members + orphan->mean * orphan->n_members) / (n_members + orphan->n_members);
	n_members += orphan->n_members;

	orphan->reassignMembersClusterID(voxels, id);	// Needed to decide which voxels are edges.

	transferMembers(orphan);				// Deletes large structures
	
	neighbor_ids.deleteVal(orphan->id);
	orphan->kill(id);						// Deletes orphans ID.
	n_neighbors = neighbor_ids.size();		// Kinda just a -- but who cares
}

bool TissueCluster3D::isMergeable(TissueCluster3D* orphan) {
	if (abs(orphan->mean - mean) < max_difference) {
		return true;
	}
	return false;
}

void TissueCluster3D::updateEdges() {

}

void TissueCluster3D::findEdges(Volume* vol) {
	for (int i = 0; i < member_indexes.size(); i++) {
		int member_index = member_indexes[i];
		Voxel* cur_voxel = &vol->voxels[member_index];
		if (isEdge(vol, indexToXYZ(member_index, vol->size), cur_voxel)) {
			//printf("Colering!")
			edge_member_indexes.push_back(member_index);
			cur_voxel->color = color;
			cur_voxel->isEdge = true;
		}
	}
}

void TissueCluster3D::transferMembers(TissueCluster3D* orphan) {
	member_indexes.insert(member_indexes.end(), orphan->member_indexes.begin(), orphan->member_indexes.end());
	orphan->member_indexes.clear();


	int* orphan_neighbors = orphan->neighbor_ids.fetch();
	for (int i = 0; i < orphan->neighbor_ids.size(); i++)
		neighbor_ids.addVal(orphan_neighbors[i]);
	orphan->neighbor_ids.clear();
	delete(orphan_neighbors);
}
void TissueCluster3D::kill(int parent_id) {
	id = parent_id;
	dead = true;
}



bool TissueCluster3D::isEdge(Volume* vol, Int3 origin, Voxel* v0) {
	for (int i = 0; i < 6; i++) {
		Int3 pos = getImmediateNeighbor(origin, i);
		if (isInVolume(pos, vol->size)) {						// Edges can on purpose not be volume-border voxels
			int neighbor_index = xyzToIndex(pos, vol->size);
			Voxel* neighbor = &vol->voxels[neighbor_index];
			if (neighbor->ignore || neighbor->cluster_id != v0->cluster_id)	// think ignore is implicit in the second comparison, as its always -1
				return true;
		}
	}
	return false;
}







