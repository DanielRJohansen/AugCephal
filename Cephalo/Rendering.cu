#include "hip/hip_runtime.h"
#include "Rendering.cuh"



Ray* RenderEngine::initRays() {
    Ray* rayptr_host = new Ray[NUM_RAYS];
    float rpd = (float)RAYS_PER_DIM;
    for (int y = 0; y < RAYS_PER_DIM; y++) {
        for (int x = 0; x < RAYS_PER_DIM; x++) {
            float x_ = 0.5 - 0.5 / rpd - x / rpd;// Shift by half increment to have
            float y_ = 0.5 - 0.5 / rpd - y / rpd;
            float d = sqrt(FOCAL_LEN * FOCAL_LEN + x_ * x_ + y_ * y_);
            rayptr_host[xyToRayIndex(y, x)] = Ray(Float3(x_, y_, FOCAL_LEN) * (1. / d));	// Yes xy is swapped, this works, so schhh!
        }
    }
    return rayptr_host;
};

__device__ __host__ int xyzToIndex(Int3 coord, Int3 size) {
    return coord.z * size.y * size.x + coord.y * size.x + coord.x;
}
__device__ inline bool isInVolume(Int3 coord, Int3 size) {
    return coord.x >= 0 && coord.y >= 0 && coord.z >= 0 && coord.x < size.x&& coord.y < size.y&& coord.z < size.z;
}


__device__ float activationFunction(float counts) {
    return 2 / (1 + powf(e, (-counts / 4.))) - 1.;
}

/*__device__ float lightSeeker(Block* volume, CudaFloat3 pos) {
    float spread = 1;
    float brightness = 1;
    for (int y = 0; y < 3; y++) {
        for (int x = 0; x < 3; x++) {

            // Upward seeking
            brightness += 1;
            for (int z = 1; z <= 64; z *= 2) {
                int vol_x = pos.x - spread + spread * x;
                int vol_y = pos.y - spread + spread * y;
                int vol_z = pos.z + z;
                if (isInVolume(Int3(vol_x, vol_y, vol_z))) {
                    int index = xyzToIndex(vol_x, vol_y, vol_z);
                    if (!volume[index].ignore) { brightness -= 1; break; }
                    //else { break; }
                }
                //else { brightness += 1; }
            }
        }
    }
    return activationFunction(brightness);
}*/

__device__ CudaFloat3 makeUnitVector(Ray* ray, CompactCam cc) {
    float x = ray->rel_unit_vector.x;
    float y = ray->rel_unit_vector.y;
    float z = ray->rel_unit_vector.z;


    // Rotate rel vector around y
    float x_y = cc.cos_pitch * x + cc.sin_pitch * z;
    float z_y = -cc.sin_pitch * x + cc.cos_pitch * z;

    // Rotate relative vector about z
    float x_z = cc.cos_yaw * x_y - cc.sin_yaw * y;
    float y_z = cc.sin_yaw * x_y + cc.cos_yaw * y;

    return CudaFloat3(x_z, y_z, z_y);
}

__global__ void testKernel(int* f) {
    *f = 99;
    return;
}


__global__ void stepKernel(Ray* rayptr, Voxel* voxels, CompactCam cc, int offset, uint8_t* image, Int3 vol_size, int* finished, bool* xyIgnores, CompactBool* CB_global, unsigned* ignores) {
    int index = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x +offset;
    *finished = 42;
    Ray ray = rayptr[index];    // This operation alone takes ~60 ms

    //CudaFloat3 unit_vector(x_z, y_z, z_y);
    CudaFloat3 unit_vector = makeUnitVector(&ray, cc);
    CudaRay cray(unit_vector * RAY_SS);
    CompactBool CB;

    Voxel* cached_voxel;
    cached_voxel = &voxels[0];  // Init Block, doesn't matter is never used before another is loaded.
    int prev_vol_index = -1;    // Impossible index


    for (int step = 100; step < RAY_STEPS; step++) {    //500

        int x = cc.origin.x + cray.step_vector.x * step;
        int y = cc.origin.y + cray.step_vector.y * step;
        int z = cc.origin.z + cray.step_vector.z * step;

        int vol_x = (int) (x + vol_size.x / 2);
        int vol_y = (int) (y + vol_size.y / 2);
        int vol_z = (int) (z + vol_size.z / 2);

        // Check if entire column is air, if then skip
        
          //  continue;
        //bool a = isInVolume(Int3(vol_x, vol_y, vol_z), vol_size);
        //Int3 b = Int3(vol_x, vol_y, vol_z);
        if (vol_x >= 0 && vol_y >= 0 && vol_z >= 0 && vol_x < vol_size.x && vol_y < vol_size.y && vol_z < vol_size.z) { // Only proceed if coordinate is within volume!
            int volume_index = xyzToIndex(Int3(vol_x, vol_y, vol_z), vol_size);

            int column_index = vol_y * vol_size.x + vol_x;
            if (CB.getBit(ignores, column_index) == 1)
                continue;

            //if (xyIgnores[vol_y * vol_size.x + vol_x])
            //    continue;

            if (vol_z == 0) {
                float remaining_alpha = 1 - cray.alpha;
                //cray.color.r = 0;
                cray.color.g += 114 * remaining_alpha;
                cray.color.b += 158 * remaining_alpha;
                break;
            }
            //if (empty_y_slices[vol_y] || empty_x_slices[vol_x]) { continue; }

            if (volume_index == prev_vol_index) {
                if (cached_voxel->ignore) { continue; }
            }
            else {
                prev_vol_index = volume_index;
                if (voxels[volume_index].ignore) { continue; }
                else { cached_voxel = &voxels[volume_index]; }
            }


            CudaColor block_color = CudaColor(cached_voxel->color.r, cached_voxel->color.g, cached_voxel->color.b);
            float brightness = 1;// lightSeeker(voxels, CudaFloat3(vol_x, vol_y, vol_z));
            block_color = block_color * brightness;
            cray.color.add(block_color * cached_voxel->alpha);
            cray.alpha += cached_voxel->alpha;
            if (cray.alpha >= 1)
                break;
        }
    }
    cray.color.cap();   //Caps each channel at 255
    image[index * 4 + 0] = (int)cray.color.r;
    image[index * 4 + 1] = (int)cray.color.g;
    image[index * 4 + 2] = (int)cray.color.b;
    image[index * 4 + 3] = 255;
}


void RenderEngine::render(sf::Texture* texture) {
    auto start = chrono::high_resolution_clock::now();
    
    CompactCam cc = CompactCam(camera->origin, camera->plane_pitch, camera->plane_yaw, camera->radius);


    hipStream_t stream[N_STREAMS];
    for (int i = 0; i < N_STREAMS; i++) {
        hipStreamCreate(&(stream[i]));
    }


    for (int i = 0; i < N_STREAMS; i++) {       // Needed because we dont want to block the GPU from other requests!
        int offset = i * stream_size;
        hipMemcpyAsync(&rayptr_device[offset], &rayptr_host[offset], ray_stream_bytes, hipMemcpyHostToDevice, stream[i]);
    }
    int* f_device;
    hipMallocManaged(&f_device, sizeof(int));
    for (int i = 0; i < N_STREAMS; i++) {
        int offset = i * stream_size;
        stepKernel << <blocks_per_sm, THREADS_PER_BLOCK, 0, stream[i] >> > (rayptr_device, voxels, cc, offset, image_device, volume->size, f_device, xyColumnIgnores, CB, compactignores);// , dev_empty_y_slices, dev_empty_x_slices);
    }

    printf("Rendering...");
    for (int i = 0; i < N_STREAMS; i++) {
        int offset = i * stream_size;
        hipMemcpyAsync(&image_host[offset * 4], &image_device[offset * 4], image_stream_bytes, hipMemcpyDeviceToHost, stream[i]);
    }

    

    hipDeviceSynchronize();
    texture->update(image_host);

    for (int i = 0; i < N_STREAMS; i++) {
        hipStreamDestroy(stream[i]);
    }

    auto stop = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::milliseconds>(stop - start);
    printf("Executed in %d ms.\n", duration);
}


